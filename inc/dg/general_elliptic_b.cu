#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cusp/print.h>

#include "backend/timer.cuh"
#include "backend/evaluation.cuh"
#include "backend/derivatives.cuh"
#include "backend/typedefs.cuh"
#include "backend/cusp_thrust_backend.h"
#include "functors.h"

#include "cg.h"
#include "elliptic.h"


//const double R_0 = 10;
//const double lx = M_PI;
//const double ly = M_PI;
//const double lz = M_PI;
//double fct(double x, double y, double z){ return sin(x-R_0)*sin(z);}
//double derivative( double x, double y, double z){return cos(x-R_0)*sin(z);}
//double laplace_fct( double x, double y, double z) { 
//    return -1./x*cos(x-R_0)*sin(z) + 2.*sin(x-R_0)*sin(z) 
//           -1./x*sin(x-R_0)*cos(z) - 2.*cos(x-R_0)*cos(z);}
//dg::bc bcx = dg::DIR;
double initial( double x, double y, double z) {return sin(0);}

const double lx = M_PI;
const double ly = M_PI;
const double lz = M_PI;
double amp = 1;
double pol( double x, double y, double z) {return 1. + amp*sin(x)*sin(z); } //must be strictly positive
double laplace_fct( double x, double y, double z) { 
    return        2.*sin(x)*sin(z)*pol(x,y, z)
       -2*pol(x,y,z)*cos(x)*cos(z)
         -amp*sin(x)*sin(x)*cos(z)*cos(z)
         -amp*cos(x)*cos(x)*sin(z)*sin(z) 
       -2*amp*sin(x)*cos(x)*cos(z)*sin(z);}
double fct(double x, double y, double z)  {      return sin(x)*sin(z);}
double derivative( double x, double y, double z){return cos(x)*sin(z);}
//double pol( double x, double y, double z) {return 1. + amp*sin(x); } //must be strictly positive
//double laplace_fct( double x, double y, double z) { 
//    return sin(x)*pol(x,y, z)
//        -amp*cos(x)*cos(x);
//}
//double fct(double x, double y, double z)  { 
//    return sin( x);}
//double derivative( double x, double y, double z){
//    return cos(x);}

int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny, Nz; 
    std::cout << "Type n, Nx, Ny and Nz\n";
    std::cin >> n >> Nx >> Ny >> Nz;
    double eps;
    std::cout << "Type epsilon! \n";
    std::cin >> eps;
    dg::Grid3d<double> grid( 0, lx, 0, ly, 0,lz, n, Nx, Ny,Nz, dg::DIR, dg::PER, dg::DIR, dg::cartesian, false);
    dg::DVec w3d = dg::create::weights( grid);
    dg::DVec v3d = dg::create::inv_weights( grid);
    dg::DVec x = dg::evaluate( initial, grid);
    dg::DVec field  =dg::evaluate( pol, grid);
    //function gets squared inside elliptic
    dg::blas1::transform( field, field, dg::SQRT<double>());

    std::cout << "Create Laplacian\n";
    t.tic();
    dg::GeneralElliptic<dg::DMatrix, dg::DVec, dg::DVec> laplace(grid, dg::not_normed, dg::centered);
    laplace.set_x( field);
    laplace.set_y( field);
    laplace.set_z( field);
    dg::DMatrix DX = dg::create::dx( grid);
    t.toc();
    std::cout<< "Creation took "<<t.diff()<<"s\n";

    dg::CG< dg::DVec > pcg( x, grid.size());

    std::cout<<"Expand right hand side\n";
    const dg::DVec solution = dg::evaluate ( fct, grid);
    const dg::DVec deriv = dg::evaluate( derivative, grid);
    dg::DVec b = dg::evaluate ( laplace_fct, grid);
    //compute W b
    dg::blas2::symv( w3d, b, b);
    
    std::cout << "For a precision of "<< eps<<" ..."<<std::endl;
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg( laplace, x, b, v3d, eps)<<std::endl;
    t.toc();
    std::cout << "... on the device took "<< t.diff()<<"s\n";
    dg::DVec  error(  solution);
    dg::blas1::axpby( 1., x,-1., error);

    double normerr = dg::blas2::dot( w3d, error);
    double norm = dg::blas2::dot( w3d, solution);
    std::cout << "L2 Norm of relative error is:               " <<sqrt( normerr/norm)<<std::endl;
    dg::blas2::gemv( DX, x, error);
    dg::blas1::axpby( 1., deriv, -1., error);
    normerr = dg::blas2::dot( w3d, error); 
    norm = dg::blas2::dot( w3d, deriv);
    std::cout << "L2 Norm of relative error in derivative is: " <<sqrt( normerr/norm)<<std::endl;
    //both function and derivative converge with order P 

    return 0;
}
