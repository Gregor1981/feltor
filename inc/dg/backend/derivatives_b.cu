#include "hip/hip_runtime.h"
#include <iostream>

#include "blas.h"
#include "derivatives.cuh"
#include "evaluation.cuh"
#include "typedefs.cuh"

#include "timer.cuh"

const double lx = 2*M_PI;
/*
double function( double x, double y, double z) { return sin(3./4.*z);}
double derivative( double x, double y, double z) { return 3./4.*cos(3./4.*z);}
dg::bc bcz = dg::DIR_NEU;
*/
double function  ( double x, double y, double z) { return sin(z);}
double derivative( double x, double y, double z) { return cos(z);}
double functionX  ( double x, double y, double z) { return sin(x);}
double derivativeX( double x, double y, double z) { return cos(x);}
dg::bc bcx = dg::DIR;

void multiply( dg::Operator<double>& op, const double* x, double* y)
{
    for( unsigned i=0; i<op.size(); i++)
    {
        y[i] = 0;
        for( unsigned j=0; j<op.size(); j++)
            y[i]+= op(i,j)*x[j];
    }
}

int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny, Nz;
    std::cout << "Note the supraconvergence!\n";
    std::cout << "Type in n, Nx, Ny and Nz!\n";
    std::cin >> n >> Nx >> Ny >> Nz;
    std::cout << "# of polynomials    " << n <<"\n";
    std::cout << "on the grid " << Nx <<" x "<<Ny<<" x "<<Nz<<"\n";
    dg::Grid3d<double> g( 0, lx, 0, lx,0, lx, n, Nx, Ny, Nz, dg::PER, dg::PER, dg::PER, dg::cartesian, false);
    dg::DMatrix dx = dg::create::dx( g, bcx, dg::normed);
    dg::DMatrix dz = dg::create::dz( g, bcx, dg::normed);
    //dg::DMatrix lxM = dg::create::laplacianM( g, bcx, dg::PER, dg::normed, dg::centered);
    const dg::DVec hv = dg::evaluate( function, g);
    const dg::DVec hX = dg::evaluate( functionX, g);
    dg::DVec hw = hv;
    dg::DVec w3d = dg::create::weights(g);
    dg::DVec v3d = dg::create::inv_weights(g);
    const dg::DVec hu = dg::evaluate( derivative, g);
    const dg::DVec huX = dg::evaluate( derivativeX, g);
    t.tic();
    dg::blas2::symv( dz, hv, hw);
    //dg::blas1::pointwiseDot( v3d, hw, hw);
    t.toc();
    std::cout << "Evaluation of dx took "<<t.diff()<<"s\n";
    dg::blas1::axpby( 1., hu, -1., hw);
    std::cout << "Distance Z to true solution: "<<sqrt(dg::blas2::dot(hw, w3d, hw))<<"\n";
    dg::blas2::symv( dx, hX, hw);
    dg::blas1::axpby( 1., huX, -1., hw);
    std::cout << "Distance X to true solution: "<<sqrt(dg::blas2::dot(hw, w3d, hw))<<"\n";
    //t.tic();
    //for( unsigned i=0; i<10; i++)
    //    dg::blas2::symv( lxM, hv, hw);
    //t.toc();
    //std::cout << "Evaluation of Lx took "<<t.diff()/10.<<"s\n";
    //dg::blas1::axpby( 1., hv, -1., hw);
    //std::cout << "Distance to true solution: "<<sqrt(dg::blas2::dot(hw, (dg::DVec)dg::create::weights(g), hw))<<"\n";
    //for periodic bc | dirichlet bc
    //n = 1 -> p = 2      2
    //n = 2 -> p = 1      1
    //n = 3 -> p = 3      3
    //n = 4 -> p = 3      3
    //n = 5 -> p = 5      5
    /*
    std::cout << "TEST VARIOUS HOST VERSIONS OF DERIVATIVE!\n";
    dg::Operator<double> forw( g.dlt().forward( ));
    dg::Operator<double> back( g.dlt().forward( ));
    dg::HMatrix DX(dx);
    const dg::HVec v = dg::evaluate( function, g);
    dg::HVec w(v);
    t.tic();
    double temp[n];
    for( unsigned i=0; i<10; i++)
        for( unsigned k=0; k<(Ny-1)*(Nx-1); k++)
        {
            multiply( forw, &v[k*n], &w[k*n]);
            multiply( back, &v[(k+1)*n], temp);
            for( unsigned j=0; j<n; j++)
                w[k*n+j] += temp[j];
        }
    t.toc();
    std::cout << "Evaluation of host derivative took "<<t.diff()/10.<<"s\n";
    t.tic();
    for( unsigned i=0; i<10; i++)
        dg::blas2::symv( DX, v, w);
    t.toc();
    std::cout << "Evaluation of host derivative took "<<t.diff()/10.<<"s\n";
    */

    return 0;
}
