#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "weights.cuh"

#include "blas.h"

double function( double x)
{
    return exp(x);
}

double function( double x, double y)
{
        return exp(x)*exp(y);
}
double function( double x, double y, double z)
{
        return exp(x)*exp(y)*exp(z);
}

const double lx = 2;
const double ly = 2;
const double lz = 2;

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;

int main()
{
    //This file tests not only the evaluation functions but also the weights
    unsigned n;
    std::cout << "Type # of polynomial coefficients ( 1, 2,...,20)!\n";
    std::cin >> n;
    unsigned Nx, Ny, Nz;
    std::cout << "# of polynomial coefficients is: "<< n<<std::endl;
    std::cout << "Type # of grid cells Nx, Ny and Nz!\n";
    std::cin >> Nx >> Ny >> Nz;
    std::cout << "# grid cells are: "<< Nx<<" x "<<Ny<<" x "<<Nz<<"\n";

    dg::Grid1d<double> g1d( 0, lx, n, Nx);
    dg::Grid2d<double> g2d( 0, lx,0, ly,n, Nx, Ny);
    dg::Grid3d<double> g3d( 0, lx,0, ly,0, lz, n, Nx, Ny, Nz,dg::PER,dg::PER,dg::PER,dg::cylindrical, false);

    //test evaluation functions
    HVec h_x = dg::evaluate( function, g1d);
    HVec h_n = dg::evaluate( function, g2d);
    HVec h_z = dg::evaluate( function, g3d);
    HVec w3d = dg::create::weights( g3d);

    //test preconditioners
    double normX = dg::blas2::dot( h_x, dg::create::weights(g1d), h_x);
    double norm2X = dg::blas2::dot( dg::create::weights(g2d), h_n);
    double norm3X = dg::blas2::dot( h_z, w3d, h_z);

    std::cout << "Square normalized 1DXnorm "<< normX <<"\n";
    double solution = (exp(4.) -exp(0))/2.;
    std::cout << "Correct square norm is    "<<solution<<std::endl;
    std::cout << "Square normalized 2DXnorm "<< norm2X<<"\n";
    double solution2 = (exp(4.)-exp(0))/2.*(exp(4.) -exp(0))/2.;
    std::cout << "Correct square norm is    "<<solution2<<std::endl;

    std::cout << "Square normalized 3DXnorm   "<< norm3X<<"\n";
    if( g3d.system() == dg::cylindrical)
        solution = (3*exp(4.)+1.)/4.;
    double solution3 = solution2*solution;
    std::cout << "Correct square norm is      "<<solution3<<std::endl;
    std::cout << "Relative 3d error is        "<<(norm3X-solution3)/solution3<<"\n";
    return 0;
} 
